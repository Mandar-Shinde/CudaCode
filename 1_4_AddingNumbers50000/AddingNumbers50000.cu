#include "hip/hip_runtime.h"
/**
* Example from CUDA Sample 7
*/

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#include <stdio.h>
#include <conio.h>

/**
* CUDA Kernel Device code
*
* Computes the vector addition of A and B into C. The 3 vectors have the same
* number of elements numElements.
*/
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}

void CUDA_SAFE_CALL(hipError_t call)
{
	hipError_t ret = call;
	if (ret != hipSuccess)
	{
		printf(" ERROR at line :%i.%d' ' %s\n", __LINE__, ret, hipGetErrorString(ret));
		exit(-1);
	}
}
/**
* Host main routine
*/
int main(void)
{
	hipError_t err = hipSuccess;
	int numElements = 50000;
	size_t size = numElements * sizeof(float);
	printf("[Vector addition of %d elements]\n", numElements);

	float *h_A = (float *)malloc(size);
	float *h_B = (float *)malloc(size);
	float *h_C = (float *)malloc(size);
	// Verify that allocations succeeded
	if (h_A == NULL || h_B == NULL || h_C == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = rand() / (float)RAND_MAX;
		h_B[i] = rand() / (float)RAND_MAX;
	}

	// Allocate momory
	float *d_A = NULL;
	float *d_B = NULL;
	float *d_C = NULL;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_A, size));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_B, size));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_C, size));

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	printf("Copy host data to device\n");
	CUDA_SAFE_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	printf("\nBlocks per Grid :%d\nThreads pre Block :%d\n", blocksPerGrid, threadsPerBlock);
	
	vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, numElements);
	
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	printf("Copy output data from the CUDA device to the host memory\n");
	err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free device and host memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);

	// Reset the device and exit
	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	err = hipDeviceReset();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Done\n");
	_getche();
	return 0;
}

