
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

/* Utility Macro : CUDA SAFE CALL */
void CUDA_SAFE_CALL(hipError_t call)
{
	hipError_t ret = call;

	switch (ret)
	{

	case hipSuccess:

		break;

	default:
	{ printf(" ERROR at line :%i.%d' ' %s\n",
		__LINE__, ret, hipGetErrorString(ret));

	exit(-1);

	break;

	}
	}
}

int main()
{
	int computCount;
	hipDeviceProp_t pro;

	CUDA_SAFE_CALL(hipGetDeviceCount(&computCount));

	for (int i = 0; i < computCount; i++)
	{
		CUDA_SAFE_CALL(hipGetDeviceProperties(&pro,i));
		printf("-------------------------------------------------");
		printf("Device ID: %d\n", computCount);
		printf("Device Name: %s\n", pro.name);
		printf("Compute :  %d.%d\n",pro.major, pro.minor);
		printf("Clock :  %d\n", pro.clockRate);
	
	}

    return 0;
}
