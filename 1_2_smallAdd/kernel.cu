#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <stdio.h> 
#include <hip/hip_runtime.h>  

__global__   void add(int a, int b, int *c)
{

	*c = a + b;

}

/* Utility Macro : CUDA SAFE CALL */


void CUDA_SAFE_CALL(hipError_t call)
{

	hipError_t ret = call;

	switch (ret)
	{
	case hipSuccess:

		break;

	default:
	{

		printf(" ERROR at line :%i.%d' ' %s\n",
			__LINE__, ret, hipGetErrorString(ret));

		exit(-1);

		break;
	}
	}
}

int main(void)
{
	int c; int *dev_c;

	CUDA_SAFE_CALL(hipMalloc((void**)& dev_c, sizeof(int)));

	add << < 1, 1 >> >(15, 13, dev_c);

	CUDA_SAFE_CALL(hipMemcpy(
		&c,
		dev_c,
		sizeof(int),
		hipMemcpyDeviceToHost));

	printf(" 15 + 13 = %d \n ", c);
	hipFree(dev_c);
	return 0;
}
