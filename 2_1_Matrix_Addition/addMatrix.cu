#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////
///  Code to add Matrix
///
/////////////////////////////////////////////////////
///  COMPILER OPTIONS
///
///
/////////////////////////////////////////////////////
///  OUTPUT
///
/////////////////////////////////////////////////////

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <memory>

enum MATRIX_TYPE{ MATRIX_INITIALIZE, MATRIX_RANDOM, MATRIX_IDENTITY };

void CUDA_SAFE_CALL(hipError_t call)
{
	hipError_t ret = call;
	if (ret != hipSuccess)
	{
		printf(" ERROR at line :%i.%d' ' %s\n", __LINE__, ret, hipGetErrorString(ret));
		exit(-1);
	}
}

__global__  void GPUAdd(float *array1, float *array2, float *result, int WIDTH)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	result[threadIdx.x] = array1[threadIdx.x] + array2[threadIdx.x];
}

void CPUAdd(float *array1, float *array2, float *result, int WIDTH)
{
	for (int i = 0; i < WIDTH; i++)
		result[i] = array1[i] + array2[i];
}

float* prepareSquareMatrix(int isz, MATRIX_TYPE typ)
{
	int matSize = isz*isz;
	float *mat = (float*)malloc(matSize* sizeof(float));

	switch (typ)
	{
	case MATRIX_INITIALIZE:
		memset(mat, 0, matSize * sizeof(float));
		break;
	case MATRIX_RANDOM:
		for (long j = 0; j<(matSize); j++)
			mat[j] = (float)rand() / (float)RAND_MAX;
		break;
	case MATRIX_IDENTITY:
		// NOT IN USE
		break;
	}
	return mat;
}



int main()
{
	// Pointer for matrix
	float *A, *B, *SUM;				// HOST
	float *cudaA, *cudaB, *cudaSUM, *cudaRET;	// DEVICE
	int msize = 1000;
	
	// Initializing matrix with data
	A = prepareSquareMatrix(msize, MATRIX_RANDOM);  // 4 X 4
	B = prepareSquareMatrix(msize, MATRIX_RANDOM);  // 4 X 4
	SUM = prepareSquareMatrix(msize, MATRIX_INITIALIZE);  // 4 X 4
	cudaRET = (float *)malloc(msize*msize*sizeof(float));

	CPUAdd(A, B, SUM, msize*msize);

	CUDA_SAFE_CALL(hipMalloc((void **)&cudaA, msize*msize*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void **)&cudaB, msize*msize*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void **)&cudaSUM, msize*msize*sizeof(float)));


	int threadsPerBlock = 256;
	int blocksPerGrid = (msize + threadsPerBlock - 1) / threadsPerBlock;
	printf("\nBlocks per Grid :%d\nThreads pre Block :%d\n", blocksPerGrid, threadsPerBlock);

	CUDA_SAFE_CALL(hipMemcpy(cudaA, A, msize*msize*sizeof(float), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(cudaB, B, msize*msize*sizeof(float), hipMemcpyHostToDevice));
	GPUAdd << <blocksPerGrid, threadsPerBlock >> >(cudaA, cudaB, cudaSUM, msize*msize);
	CUDA_SAFE_CALL(hipMemcpy(cudaRET, cudaSUM, msize*msize, hipMemcpyDeviceToHost));
	
	hipFree(cudaA);
	hipFree(cudaB);
	hipFree(cudaSUM);
	free(A);
	free(B);
	free(SUM);

    return 0;
}
