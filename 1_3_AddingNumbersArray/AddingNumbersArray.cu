#include "hip/hip_runtime.h"

/////////////////////////////////////////////////////
///  Code to listdown attached compute device
///
/////////////////////////////////////////////////////
///  COMPILER OPTIONS
///
///  C:\Users\mandar\Documents\GitHub\CudaCode\1_2_AddingNumber>"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v7.0\bin\nvcc.exe" - gencode = arch = compute_20, code = \"sm_20,compute_20\" --use-local-env --cl-version 2013 -ccbin "C:\Program Files(x86)\Microsoft Visual Studio 12.0\VC\bin"  -I"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v7.0\include" -I"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v7.0\include"  -G   --keep-dir Debug -maxrregcount=0  --machine 32 --compile -cudart static  -g   -DWIN32 -D_DEBUG -D_CONSOLE -D_MBCS -Xcompiler " / EHsc / W3 / nologo / Od / Zi / RTC1 / MDd  " -o Debug\AddingNumber.cu.obj "C:\Users\mandar\Documents\GitHub\CudaCode\1_2_AddingNumber\AddingNumber.cu" 
///
/////////////////////////////////////////////////////
///  OUTPUT
///
///  15 + 13 = 28
/////////////////////////////////////////////////////
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <stdio.h>
#include <conio.h>


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void CUDA_SAFE_CALL(hipError_t call)
{
	hipError_t ret = call;
	if (ret != hipSuccess)
	{
		printf(" ERROR at line :%i.%d' ' %s\n", __LINE__, ret, hipGetErrorString(ret));
		exit(-1);
	}
}

int main()
{
	printf("Adding Using Cuda \n ");
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	CUDA_SAFE_CALL(addWithCuda(c, a, b, arraySize));

	printf("\n\n    [1,2,3,4,5]  +  [10,20,30,40,50] \n   = [%d, %d, %d, %d, %d]\n",c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	CUDA_SAFE_CALL(hipDeviceReset());
	
	_getche();
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus=hipSuccess;

	// Choose which GPU to run on, change this on a multi-GPU system.
	CUDA_SAFE_CALL(hipSetDevice(0));
	
	// Allocate GPU buffers for three vectors (two input, one output)    .
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_c, size * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_a, size * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&dev_b, size * sizeof(int)));

	// Copy input vectors from host memory to GPU buffers.
	CUDA_SAFE_CALL(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));	
	CUDA_SAFE_CALL(hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice));
	
	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	CUDA_SAFE_CALL(hipGetLastError());
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	// Copy output vector from GPU buffer to host memory.
	CUDA_SAFE_CALL(hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost));

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}